#include "hip/hip_runtime.h"
/*
 * CUDAUtility.cpp
 *
 *  Created on: 13/10/2016
 *      Author: abian
 */

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <random>

#include "CUDAUtility.cuh"

#define min(a,b) (a<b?a:b);

__device__ size_t sampleIDsPitch_const;
__device__ size_t inbagCountsPitch_const;

/**
 * @brief Algorithm bootstrap not weighted with replacement for n tree. It will generate N bootstrap samples with the objective to generate a bootstrap sample for echa tree.
 *
 * @param nTree number of bootstrap samples that it have to generate.
 * @param nSamples number of samples of the original dataset.
 * @param sampleFraction number of fraction for each sample.
 * @param seed a array with random seed for each tree for to generate random number. The seed length is equal to nTree.
 * @param sampleIDs (output) bootstrap sample for each tree.
 * @param inbagCounts (output) histogram of each bootstrap sample.
 */
__global__ void bootstrap_kernel(size_t nTree, size_t nSamples, double sampleFraction, uint* seed, size_t* sampleIDs,
    uint* inbagCounts){
  int tid = threadIdx.x;
  int offset = blockDim.x;

  /*Generating a random number in a specific ranger:
    1- Use CURAND to generate a uniform distribution between 0.0 and 1.0
    2- Then multiply this by the desired range (largest value - smallest value + 0.999999).
    3- Then add the offset (+ smallest value).
    4- Then truncate to an integer.
  */
  hiprandState state;
  hiprand_init(seed[blockIdx.x], tid + blockIdx.x * blockDim.x, 0, &state);
  while(tid < nSamples*sampleFraction){
    float randf = hiprand_uniform(&state);
    randf *= ((nSamples - 1) - 0) + 0.999999;
    randf += 0;
    int rand = (int)truncf(randf);

    //Row sample
    size_t* rSample = (size_t *)((char *)sampleIDs + blockIdx.x*sampleIDsPitch_const);
    rSample[tid] = rand;

    //Row count
    uint* rCount = (uint *)((char *)inbagCounts + blockIdx.x*inbagCountsPitch_const);
    atomicAdd(&(rCount[rand]), 1);

    tid += offset;
  }
}

CUDAUtility::CUDAUtility() : maxThreadsPerBlock(512) {}

CUDAUtility::~CUDAUtility() {}

CUDAUtility& CUDAUtility::getInstance(){
	static CUDAUtility instance;
	return instance;
}

void CUDAUtility::bootstrap(size_t nSamples, double sampleFraction, size_t nTree, std::vector<uint>seeds,
    std::vector<std::vector<size_t>>& samplesIDs, std::vector<std::vector<uint>>& inbagCounts){

  //Host var
  size_t *host_sampleIDs;
  uint *host_inbagCounts;
  host_sampleIDs = (size_t *)malloc((int)(nSamples * sampleFraction * nTree) * sizeof(size_t));
  host_inbagCounts = (uint *)malloc(nSamples * nTree * sizeof(int));
  //How i use memory in 2D, I need the pitch
  size_t host_sampleIDs_pitch = nSamples * sampleFraction * sizeof(size_t);
  size_t host_inbagCounts_pitch = nSamples * sizeof(int);

  //Device var
  size_t *dev_sampleIDs;
  uint *dev_inbagCounts, *dev_seed;
  size_t dev_sampleIDs_pitch, dev_inbagCounts_pitch;

  hipMallocPitch((void **)&dev_sampleIDs, &dev_sampleIDs_pitch, (int)(nSamples * sampleFraction) * sizeof(size_t), nTree);
  hipMallocPitch((void **)&dev_inbagCounts, &dev_inbagCounts_pitch, nSamples * sizeof(int), nTree);
  hipMalloc((void **)&dev_seed, nTree * sizeof(int));
  hipMemcpy(dev_seed, seeds.data(), nTree * sizeof(int), hipMemcpyHostToDevice);

  //Initialize the histogram of inbag samples
  hipMemset2D(dev_inbagCounts, dev_inbagCounts_pitch, 0, nSamples * sizeof(int), nTree);

  hipMemcpyToSymbol(HIP_SYMBOL(sampleIDsPitch_const), &dev_sampleIDs_pitch, sizeof(size_t));
  hipMemcpyToSymbol(HIP_SYMBOL(inbagCountsPitch_const), &dev_inbagCounts_pitch, sizeof(size_t));

  int threadsPerBlock = min(nSamples, maxThreadsPerBlock);
  bootstrap_kernel<<<nTree,threadsPerBlock>>>(nTree, nSamples, sampleFraction, dev_seed, dev_sampleIDs,
      dev_inbagCounts);

  hipMemcpy2D(host_sampleIDs, host_sampleIDs_pitch, dev_sampleIDs, dev_sampleIDs_pitch, host_sampleIDs_pitch,
      nTree, hipMemcpyDeviceToHost);
  hipMemcpy2D(host_inbagCounts, host_inbagCounts_pitch, dev_inbagCounts, dev_inbagCounts_pitch,
      host_inbagCounts_pitch, nTree, hipMemcpyDeviceToHost);

  arrayToVector(samplesIDs, host_sampleIDs, host_sampleIDs_pitch/sizeof(size_t), nTree);
  arrayToVector(inbagCounts, host_inbagCounts, host_inbagCounts_pitch/sizeof(int), nTree);

  free(host_sampleIDs);
  free(host_inbagCounts);
  hipFree(dev_sampleIDs);
  hipFree(dev_inbagCounts);
  hipFree(dev_seed);

  return;
}

template<typename T>
void CUDAUtility::arrayToVector(std::vector<std::vector<T>> &result, T *array, size_t width, size_t height){

  for (int i=0; i<height; ++i){
    std::vector<T> row ( &array[i*width], &array[(i+1)*width] );
    result.push_back(row);
  }

  return;
}
